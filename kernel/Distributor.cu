#include "hip/hip_runtime.h"
#include "Distributor.h"

__global__ void Distributor(UpdateCacheArea *d_cache_update, UpdateType *d_buffer_update, Config *d_config, SecIndex *d_secindex, Grid *d_grid)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int anchor;

	anchor = tid;
	while (anchor < d_config->block_size_update)
	{
		UpdateType ins_update = d_buffer_update[anchor];

	   	SIEntry p_sie = d_secindex->index[ins_update.oid];
	   	int old_cell_id, new_cell_id;
	   	old_cell_id = p_sie.idx_cell;

	    int i = ins_update.x;
	   	int j = ins_update.y;
	   	new_cell_id = i + j * d_config->edge_cell_num;

	   	if (new_cell_id == old_cell_id)
	   	{
	    	if (d_cache_update->sum_f[old_cell_id] < d_config->len_seg_cache_update)
	    	{
                int cnt_f = __iAtomicAdd(&(d_cache_update->sum_f[old_cell_id]), 1);      //very important
		        d_cache_update->mtx_fresh[old_cell_id * d_config->len_seg_cache_update + cnt_f] = ins_update;
	    	}
	   	}
	   	else
	   	{
	    	if (d_cache_update->sum_d[old_cell_id] < d_config->len_seg_cache_update && \
	    			d_cache_update->sum_i[new_cell_id] < d_config->len_seg_cache_update)
		   	{
                int cnt_d = __iAtomicAdd(&(d_cache_update->sum_d[old_cell_id]), 1);
		       	d_cache_update->mtx_delete[old_cell_id * d_config->len_seg_cache_update + cnt_d] = ins_update.oid;
			
                int cnt_i = __iAtomicAdd(&(d_cache_update->sum_i[new_cell_id]), 1);
		        d_cache_update->mtx_insert[new_cell_id * d_config->len_seg_cache_update + cnt_i] = ins_update;
	    	}
	   	}
		anchor += gridDim.x * blockDim.x;
	}
	__syncthreads();    

	if (tid == 0)
    	printf("Ending Distributing Update Kernel...\n");

}

