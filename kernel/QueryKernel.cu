#include "hip/hip_runtime.h"
#include "QueryKernel.h"

__global__ void QueryDispatch(QueryCacheArea *d_cache_query, QueryType *d_buffer_query, Config *d_config)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int anchor = tid;
	while (anchor < d_config->block_size_query)
	{
		QueryType que = d_buffer_query[anchor];
		int rs = que.minY;
		int re = que.maxY;
		int cs = que.minX;
		int ce = que.maxX;
		for (int i = rs; i <= re; i++)
		{
			for (int j = cs; j <= ce; j++)
			{
				int cid = i * d_config->edge_cell_num + j;
				if (i == rs || i == re || j == cs || j == ce)
				{
					d_cache_query->partial[anchor * d_cache_query->seg2 + d_cache_query->cnt2[anchor]] = cid;
					d_cache_query->cnt2[anchor]++;
				}
				else
				{
					d_cache_query->total[anchor * d_cache_query->seg1 + d_cache_query->cnt1[anchor]] = cid;
					d_cache_query->cnt1[anchor]++;
				}
			}
		}
		anchor += gridDim.x * blockDim.x;
	}
	__syncthreads();
	if (tid == 0)
		printf("Ending QueryDispatch...\n");
}

__device__ int CountObject(int rs, int re, int cs, int ce, Config *config, Grid *grid)
{
	int res = 0;
	for (int i = rs; i <= re; i++)
	{
		for (int j = cs; j <= ce; j++)
		{
			int cid = i * config->edge_cell_num + j;
			int nB = grid->arr_cell[cid].nB;
			if (nB > 0)
				res = res + grid->arr_cell[cid].head->nO + (nB - 1) * config->max_bucket_len;
		}
	}
	return res;
}

__device__ void QueryWrite(QueryOut *query_out, QueryCacheArea *cache_query, QueryType que, Grid *grid, int anchor)  // 采用一个virtual warp处理一个查询；
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int woff = tid % 8;
	int k = woff;
	while (k < cache_query->cnt1[anchor])
	{
		int cid = cache_query->total[anchor * cache_query->seg1 + k];
		Bucket *cur_bkt = grid->arr_cell[cid].head;
		while (cur_bkt != NULL)
		{
			for (int j = 0; j < cur_bkt->nO; j++)
			{
				int pos;
				if (query_out->cnt[anchor] < query_out->seg)
                    pos = __iAtomicAdd(&(query_out->cnt[anchor]), 1);
				else
					pos = query_out->cnt[anchor];
				query_out->obj_id[anchor * query_out->seg + pos] = cur_bkt->arr_obj[j].oid;
			}
			cur_bkt = cur_bkt->nxt;
		}
		k += 8;
	}
	cache_query->cnt1[anchor] = 0;

	k = woff;
	while (k < cache_query->cnt2[anchor])
	{
		int cid = cache_query->partial[anchor * cache_query->seg1 + k];
		Bucket *cur_bkt = grid->arr_cell[cid].head;
		while (cur_bkt != NULL)
		{
			for (int j = 0; j < cur_bkt->nO; j++)
			{
				if (cur_bkt->arr_obj[j].x >= que.minX && cur_bkt->arr_obj[j].x < que.maxX && cur_bkt->arr_obj[j].y >= que.minY && cur_bkt->arr_obj[j].y < que.maxY)
				{
			    	int pos;
			    	if (query_out->cnt[anchor] < query_out->seg)
			    		pos = __iAtomicAdd(&(query_out->cnt[anchor]), 1);
		    		else
			    		pos = query_out->cnt[anchor];
			    	query_out->obj_id[anchor * query_out->seg + pos] = cur_bkt->arr_obj[j].oid;
		    	}
			}
			cur_bkt = cur_bkt->nxt;
		}
		k += 8;
	}
	cache_query->cnt2[anchor] = 0;
}


__global__ void QueryKernel(QueryOut *d_query_out, QueryCacheArea *d_cache_query, QueryType *d_buffer_query, Config *d_config, Grid *d_grid)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int wid = tid / 8; 
	int stride = blockDim.x * gridDim.x / 8;
	int anchor = wid;
	while (anchor < d_config->block_size_query)
	{
		QueryType que = d_buffer_query[anchor];
	//	DynamicParallelism(d_query_out, d_cache_query, que, d_config, d_grid, anchor, Avg);
        QueryWrite(d_query_out, d_cache_query, que, d_grid, anchor);
		anchor += stride;
	}
	if (tid == 0)
		printf("Ending QueryKernel...\n");
}

/*__device__ void DynamicParallelism(QueryOut *query_out, QueryCacheArea *cache_query, QueryType que, Config *config, Grid *grid, int anchor, int Avg)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int woff = tid % 4;
	int rs = que.minY;
	int re = que.maxY;
	int cs = que.minX;
	int ce = que.maxX;
	int Cur = CountObject(rs, re, cs, ce, config, grid);
	if (Cur > Avg * 2)
	{
		if (woff == 0)
			DynamicKernel<<<1, 8>>>(query_out, cache_query, que, grid, anchor);
	}
}

__global__ void DynamicKernel(QueryOut *query_out, QueryCacheArea *cache_query, QueryType que, Grid *grid, int anchor)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int k = tid;
	while (k < cache_query->cnt1[anchor])
	{
		int cid = cache_query->total[anchor * cache_query->seg1 + k];
		Bucket *cur_bkt = grid->arr_cell[cid].head;
		while (cur_bkt != NULL)
		{
			for (int j = 0; j < cur_bkt->nO; j++)
			{
				int pos;
				if (query_out->cnt[anchor] < query_out->seg)
					pos = __iAtomicAdd(&(query_out->cnt[anchor]), 1);
				else
					pos = query_out->cnt[anchor];
				query_out->obj_id[anchor * query_out->seg + pos] = cur_bkt->arr_obj[j].oid;
			}
			cur_bkt = cur_bkt->nxt;
		}
		k += gridDim.x * blockDim.x;
	}
    cache_query->cnt1[anchor] = 0;

	k = tid;
	while (k < cache_query->cnt2[anchor])
	{
		int cid = cache_query->partial[anchor * cache_query->seg2 + k];
		Bucket *cur_bkt = grid->arr_cell[cid].head;
		while (cur_bkt != NULL)
		{
	    	for (int j = 0; j < cur_bkt->nO; j++)
	    	{
	    		if (cur_bkt->arr_obj[j].x >= que.minX && cur_bkt->arr_obj[j].x < que.maxX && cur_bkt->arr_obj[j].y >= que.minY && cur_bkt->arr_obj[j].y < que.maxY)
	    		{
	    			int pos;
		    		if (query_out->cnt[anchor] < query_out->seg)
		    			pos = __iAtomicAdd(&(query_out->cnt[anchor]), 1);
		    		else
			    		pos = query_out->cnt[anchor];
			    	query_out->obj_id[anchor * query_out->seg + pos] = cur_bkt->arr_obj[j].oid;
		    	}
			}
			cur_bkt = cur_bkt->nxt;
		}
		k += gridDim.x * blockDim.x;
	}
}     */

/*__global__ void QueryKernel(QueryOut *d_query_out, QueryCacheArea *d_cache_query, QueryType * d_buffer_query, Config *d_config, Grid *d_grid, int Avg)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int anchor = tid;
	while (anchor < d_config->block_size_query)
	{
		QueryType que = d_buffer_query[anchor];
		int rs = que.minY;
		int re = que.maxY;
		int cs = que.minX;
		int ce = que.maxX;
		int Cur = CountObject(rs, re, cs, ce, d_config, d_grid);
        if (Cur > Avg)
			DynamicParallelism<<<1, 4 >>>(d_query_out, d_cache_query, d_grid, que, anchor);
		else
		{
			for (int i = 0; i < d_cache_query->cnt1[anchor]; i++)
			{
				int cid = d_cache_query->total[anchor * d_cache_query->seg1 + i];
				Bucket *cur_bkt = d_grid->arr_cell[cid].head;
				while (cur_bkt != NULL)
				{
					for (int j = 0; j < cur_bkt->nO; j++)
					{
						d_query_out->obj_id[anchor * d_query_out->seg + d_query_out->cnt[anchor]] = cur_bkt->arr_obj[j].oid;
						if (d_query_out->cnt[anchor] < d_query_out->seg)
							 ++d_query_out->cnt[anchor];
					}
					cur_bkt = cur_bkt->nxt;
				}
			}
			d_cache_query->cnt1[anchor] = 0;

			for (int i = 0; i < d_cache_query->cnt2[anchor]; i++)
			{
				int cid = d_cache_query->partial[anchor * d_cache_query->seg2 + i];
				Bucket *cur_bkt = d_grid->arr_cell[cid].head;
				while (cur_bkt != NULL)
				{
					for (int j = 0; j < cur_bkt->nO; j++)
					{
						if (cur_bkt->arr_obj[j].x >= que.minX && cur_bkt->arr_obj[j].x < que.maxX && cur_bkt->arr_obj[j].y >= que.minY && cur_bkt->arr_obj[j].y < que.maxY)
						{
					    	d_query_out->obj_id[anchor * d_query_out->seg + d_query_out->cnt[anchor]] = cur_bkt->arr_obj[j].oid;
					    	if (d_query_out->cnt[anchor] < d_query_out->seg)
						    	 ++d_query_out->cnt[anchor];
						}
					}
					cur_bkt = cur_bkt->nxt;
				}
			}
			d_cache_query->cnt2[anchor] = 0;  
		}    
		anchor += gridDim.x * blockDim.x;
	}  
	if (tid == 0)
		printf("Ending QueryKernel...\n");
}   */

