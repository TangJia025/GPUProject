#include "hip/hip_runtime.h"
#include "UpdateKernel.h"

__global__ void UpdateKernel(UpdateCacheArea *d_cache_update, Config *d_config, Grid *d_grid, SecIndex *d_secindex, Bktpond *d_bkt_pond, Buffer *d_free_bkt_id, int *offset1, int *offset2)
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int anchor;

	anchor = tid;
	while (anchor < d_config->edge_cell_num * d_config->edge_cell_num)
	{
		for (int i = 0; i < d_cache_update->sum_f[anchor]; i++)
		{
			UpdateType ins_update = d_cache_update->mtx_fresh[anchor * d_config->len_seg_cache_update + i];
            d_cache_update->mtx_fresh[anchor * d_config->len_seg_cache_update + i].oid = -1;
            d_cache_update->mtx_fresh[anchor * d_config->len_seg_cache_update + i].x = -1;
            d_cache_update->mtx_fresh[anchor * d_config->len_seg_cache_update + i].y = -1;
            d_cache_update->mtx_fresh[anchor * d_config->len_seg_cache_update + i].vx = -1;
            d_cache_update->mtx_fresh[anchor * d_config->len_seg_cache_update + i].vy = -1;
            d_cache_update->mtx_fresh[anchor * d_config->len_seg_cache_update + i].ts = -1;
			int oid = ins_update.oid;
            int idx_bkt = d_secindex->index[oid].idx_bkt;
			int idx_obj = d_secindex->index[oid].idx_obj;
			Bucket *cur_bkt = d_grid->arr_cell[anchor].head;
			int j = 0;
			while (j++ < d_grid->arr_cell[anchor].nB - idx_bkt -1) 	cur_bkt = cur_bkt->nxt;
			ObjBox *cur_obj = &(cur_bkt->arr_obj[idx_obj]);
			cur_obj->x = ins_update.x;
			cur_obj->y = ins_update.y;
			cur_obj->vx = ins_update.vx;
			cur_obj->vy = ins_update.vy;
			cur_obj->ts = ins_update.ts;
		}
        d_cache_update->sum_f[anchor] = 0;      
		anchor += blockDim.x * gridDim.x;
	}       
	__syncthreads();
    if (threadIdx.x == 0)
	{
		__iAtomicAdd(offset1, 1);
	}
	__syncthreads();
	while (*offset1 != gridDim.x);
	if (tid == 0)
		printf("Ending Update Fresh Kernel...\n");
    
	anchor = tid;
	while (anchor < d_config->edge_cell_num * d_config->edge_cell_num)
	{
    	for (int i = 0; i < d_cache_update->sum_d[anchor]; i++)
    	{
	    	int oid = d_cache_update->mtx_delete[anchor * d_config->len_seg_cache_update + i];
	    	d_cache_update->mtx_delete[anchor * d_config->len_seg_cache_update + i] = -1;
	    	int idx_bkt = d_secindex->index[oid].idx_bkt;
	    	int idx_obj = d_secindex->index[oid].idx_obj;
            ObjBox *last_obj = &(d_grid->arr_cell[anchor].head->arr_obj[d_grid->arr_cell[anchor].head->nO-1]);

    		Bucket *cur_bkt = d_grid->arr_cell[anchor].head;
	    	int j = 0; 
	    	while (j++ < d_grid->arr_cell[anchor].nB - idx_bkt - 1) cur_bkt = cur_bkt->nxt;
	    	ObjBox *cur_obj = &(cur_bkt->arr_obj[idx_obj]);
	    	if (cur_obj->oid == last_obj->oid)
	    	{
	        	if (d_grid->arr_cell[anchor].head->nO >= 2)
	        	{
			    	last_obj->oid = -1;
			    	last_obj->x = -1;
			    	last_obj->y = -1;
			    	last_obj->vx = -1;
			    	last_obj->vy = -1;
			    	last_obj->ts = -1;
			    	d_grid->arr_cell[anchor].head->nO--;
		    	}
		    	else
		    	{
			    	int rear = __iAtomicAdd(&(d_free_bkt_id->rear), 1);
			    	rear %= d_free_bkt_id->len;
			      	int bid = d_grid->arr_cell[anchor].head->bid;
	    	      	d_free_bkt_id->bkt_id[rear] = bid;
                    d_grid->arr_cell[anchor].head = d_bkt_pond->arr_bkt[bid].nxt;
		         	d_bkt_pond->arr_bkt[bid].nxt = NULL;
 
     			   	d_bkt_pond->arr_bkt[bid].arr_obj[0].oid = -1;
	    		   	d_bkt_pond->arr_bkt[bid].arr_obj[0].x = -1;
		    	   	d_bkt_pond->arr_bkt[bid].arr_obj[0].y = -1;
		    	   	d_bkt_pond->arr_bkt[bid].arr_obj[0].vx = -1;
			      	d_bkt_pond->arr_bkt[bid].arr_obj[0].vy = -1;
			      	d_bkt_pond->arr_bkt[bid].arr_obj[0].ts = -1;
			      	d_bkt_pond->arr_bkt[bid].nO = 0;
			     	d_grid->arr_cell[anchor].nB--;  
		    	}
	    	}
	    	else
	    	{
                cur_obj->oid = last_obj->oid;
		    	cur_obj->x = last_obj->x;
		    	cur_obj->y = last_obj->y;
		    	cur_obj->vx = last_obj->vx;
		    	cur_obj->vy = last_obj->vy;
		    	cur_obj->ts = last_obj->ts;
		    	d_secindex->index[last_obj->oid].idx_bkt = idx_bkt;
			    d_secindex->index[last_obj->oid].idx_obj = idx_obj;
		    	if (d_grid->arr_cell[anchor].head->nO >= 2)
		    	{
		    		last_obj->oid = -1;
			    	last_obj->x = -1;
			    	last_obj->y = -1;
			    	last_obj->vx = -1;
			    	last_obj->vy = -1;
			    	last_obj->ts = -1;
			    	d_grid->arr_cell[anchor].head->nO--;
		    	}
		    	else
		    	{
		    		int rear = __iAtomicAdd(&(d_free_bkt_id->rear), 1);
			    	rear %= d_free_bkt_id->len;
			      	int bid = d_grid->arr_cell[anchor].head->bid;
	    	     	d_free_bkt_id->bkt_id[rear] = bid;
                    d_grid->arr_cell[anchor].head = d_bkt_pond->arr_bkt[bid].nxt;
			     	d_bkt_pond->arr_bkt[bid].nxt = NULL;
 
			      	d_bkt_pond->arr_bkt[bid].arr_obj[0].oid = -1;
			      	d_bkt_pond->arr_bkt[bid].arr_obj[0].x = -1;
			      	d_bkt_pond->arr_bkt[bid].arr_obj[0].y = -1;
			      	d_bkt_pond->arr_bkt[bid].arr_obj[0].vx = -1;
			      	d_bkt_pond->arr_bkt[bid].arr_obj[0].vy = -1;
			      	d_bkt_pond->arr_bkt[bid].arr_obj[0].ts = -1;
			      	d_bkt_pond->arr_bkt[bid].nO = 0;
			     	d_grid->arr_cell[anchor].nB--;  
		    	}
	    	}
    	}
        d_cache_update->sum_d[anchor] = 0;    
        anchor += blockDim.x * gridDim.x;
    }    
	__syncthreads();
	if (threadIdx.x == 0)
	{
		__iAtomicAdd(offset2, 1);
	}
	__syncthreads();
    while (*offset2 != gridDim.x);
	if (tid == 0)
		printf("Ending Update Delete Kernel...\n");

	anchor = tid;
	while (anchor < d_config->edge_cell_num * d_config->edge_cell_num)
	{
		for (int i = 0; i < d_cache_update->sum_i[anchor]; i++)
		{
			UpdateType ins_update = d_cache_update->mtx_insert[anchor * d_config->len_seg_cache_update + i];
            d_cache_update->mtx_insert[anchor * d_config->len_seg_cache_update + i].oid = -1;
            d_cache_update->mtx_insert[anchor * d_config->len_seg_cache_update + i].x = -1;
            d_cache_update->mtx_insert[anchor * d_config->len_seg_cache_update + i].y = -1;
            d_cache_update->mtx_insert[anchor * d_config->len_seg_cache_update + i].vx = -1;
            d_cache_update->mtx_insert[anchor * d_config->len_seg_cache_update + i].vy = -1;
            d_cache_update->mtx_insert[anchor * d_config->len_seg_cache_update + i].ts = -1;

            if (d_grid->arr_cell[anchor].head == NULL)
			{
				int front = __iAtomicAdd(&(d_free_bkt_id->front), 1);
				front %= d_free_bkt_id->len;
				int bid = d_free_bkt_id->bkt_id[front];
				d_grid->arr_cell[anchor].head = &(d_bkt_pond->arr_bkt[bid]);
				d_grid->arr_cell[anchor].nB++;
		    }
			else if (d_grid->arr_cell[anchor].head->nO >= d_config->max_bucket_len)
			{
     			int front = __iAtomicAdd(&(d_free_bkt_id->front), 1);
				front %= d_free_bkt_id->len;
				int bid = d_free_bkt_id->bkt_id[front];
				int h_bid = d_grid->arr_cell[anchor].head->bid;
				d_grid->arr_cell[anchor].head = &(d_bkt_pond->arr_bkt[bid]);
				d_bkt_pond->arr_bkt[bid].nxt = &(d_bkt_pond->arr_bkt[h_bid]);
				d_grid->arr_cell[anchor].nB++;
			}     
			d_grid->arr_cell[anchor].head->arr_obj[d_grid->arr_cell[anchor].head->nO].oid = ins_update.oid;
			d_grid->arr_cell[anchor].head->arr_obj[d_grid->arr_cell[anchor].head->nO].x = ins_update.x;
			d_grid->arr_cell[anchor].head->arr_obj[d_grid->arr_cell[anchor].head->nO].y = ins_update.y;
			d_grid->arr_cell[anchor].head->arr_obj[d_grid->arr_cell[anchor].head->nO].vx = ins_update.x;
			d_grid->arr_cell[anchor].head->arr_obj[d_grid->arr_cell[anchor].head->nO].vy = ins_update.y;
			d_grid->arr_cell[anchor].head->arr_obj[d_grid->arr_cell[anchor].head->nO].ts = ins_update.ts;   
			d_grid->arr_cell[anchor].head->nO++;        

            d_secindex->index[ins_update.oid].idx_cell = anchor;
			d_secindex->index[ins_update.oid].idx_bkt = d_grid->arr_cell[anchor].nB-1;   
			d_secindex->index[ins_update.oid].idx_obj = d_grid->arr_cell[anchor].head->nO-1;    
		}
        d_cache_update->sum_i[anchor] = 0;       
		anchor += gridDim.x * blockDim.x;   
	}       
	__syncthreads();
	if (tid == 0)
		printf("Ending Update Insert Kernel...\n");
} 


