#include <iostream>
#include <stdio.h>

#include "header/Config.h"
#include "header/Bucket.h"
#include "header/Buffer.h"
#include "header/SecIndex.h"
#include "header/Grid.h"
#include "header/RandNumber.h"
#include "header/UpdateCacheArea.h"
#include "header/QueryCacheArea.h"
#include "header/QueryOut.h"

#include "kernel/Distributor.h"
#include "kernel/UpdateKernel.h"
#include "kernel/QueryKernel.h"

using namespace std;

Config *config = NULL;
Bktpond *bkt_pond = NULL;
Buffer *free_bkt_id = NULL;
Grid *grid = NULL;
SecIndex *secindex = NULL;
UpdateBuffer *orig_buffer_update = NULL;
QueryBuffer *orig_buffer_query = NULL;
UpdateCacheArea *cache_update = NULL;
QueryCacheArea *cache_query;  // 存储每个查询请求的部分 和全部 查询Cell号；
QueryOut *query_out = NULL;
FILE *fp1, *fp2;  // 分别表示 更新文件句柄、 查询文件句柄；
hipStream_t stream1, stream2, stream3;
hipEvent_t start1, stop1;
float elapsedTime;
int *offset1, *offset2;

void CtorConfig();
void CtorBktpond();
void CtorBuffer();
void CtorGridandSecIndex();
void CtorUpdateBuffer();
void WriteUpdateBuffer();
void CtorQueryBuffer();
void WriteQueryBuffer();
void CtorCacheUpdate();
void CtorCacheQuery();
void CtorQueryOut();
void Dtor();

int main()
{
    hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3); 

	hipEventCreate(&start1);
	hipEventCreate(&stop1);  

	cout << " ****************** " << endl;
	CtorConfig(); // 创建参数文件config;
    CtorBktpond(); // 创建内存池Bktpond;
	CtorBuffer();  // 创建Bucket号环形缓冲区；
	CtorGridandSecIndex(); // 创建区域索引Grid 和 二级索引SecIndex;
	CtorUpdateBuffer();  // 创建更新请求缓冲区orig_buffer_update;
	CtorQueryBuffer();  // 创建查询请求缓冲区orig_buffer_query;
	CtorCacheUpdate();
	CtorCacheQuery();
    CtorQueryOut();  // 创建查询输出缓冲区query_out;
	cout << " ****************** " << endl;
    hipMallocManaged(&offset1, sizeof(int));
	hipMallocManaged(&offset2, sizeof(int));
    
	if (config->gaussian_data == 1)
	{
		cout << "Reading Gaussian Data..." << endl;
		fp1 = fopen("generator/gaussian/update.txt", "r");
		fp2 = fopen("generator/gaussian/query.txt", "r");
		if (fp1 == NULL || fp2 == NULL)
		{
			cout << "Opening Gaussian File Error..." << endl;
			return 1;
		}
	}
	else
	{
		cout << "Reading Uniform Data..." << endl;
		fp1 = fopen("generator/uniform/update.txt", "r");
		fp2 = fopen("generator/uniform/query.txt", "r");
		if (fp1 == NULL || fp2 == NULL)
		{
			cout << "Opening Uniform File Error..." << endl;
			return 1;
		}
	}
	WriteUpdateBuffer();
	WriteQueryBuffer();
    hipEventRecord(start1, 0);
//	int Avg = config->max_obj_num * config->query_width_rate * config->query_width_rate;
	for (int i = 0; i < 25; i++)
	{
	cout << "Round " << i << endl;
/*	*offset1 = *offset2 = 0;
    Distributor<<<3, 1024, 0, stream1>>>\
         	(cache_update, orig_buffer_update->buffer_update + i * config->block_size_update, config, secindex, grid);
   	hipStreamSynchronize(stream1);
	UpdateKernel<<<3, 1024, 0, stream1>>>\
		    (cache_update, config, grid, secindex, bkt_pond, free_bkt_id, offset1, offset2);
	hipStreamSynchronize(stream1);    */
	QueryDispatch<<<10, 1024, 0, stream1>>>\
		    (cache_query, orig_buffer_query->buffer_query + i * config->block_size_query, config);
	hipStreamSynchronize(stream1);
	QueryKernel<<<10, 1024, 0, stream1>>>\
			(query_out, cache_query, orig_buffer_query->buffer_query + i * config->block_size_query, config, grid);
	hipStreamSynchronize(stream1); 
	for (int j = 0; j < query_out->query_num; j++) query_out->cnt[j] = 0; // 算是给QueryOut清空；   
	}
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&elapsedTime, start1, stop1);
    cout << "Time taken: " << elapsedTime << "ms" << endl;      

	Dtor();
	return 0;
}

void CtorConfig()
{
    hipMallocManaged(&config, sizeof(Config));

	config->edge_cell_num = 128;
	config->gaussian_data = 1;
	config->hotspot_num = 100;
	config->max_bucket_len = 1000;
	config->bucket_num = 22000;
	config->query_width_rate = 0.02;

	config->region_xmin = 0;
	config->region_xmax = config->edge_cell_num - 1;
	config->region_ymin = 0;
	config->region_ymax = config->edge_cell_num - 1;
	config->max_obj_num = 10000000;
	config->max_update_num = 40000000;
	config->max_query_num = 4000000; 
	config->block_size_update = 400000;
	config->block_size_query = 40000;
	config->block_analysis_num = 3;
	config->block_update_num = 8;
	config->block_query_num = 8;
	config->thread_analysis_num = 1024; 
	config->len_seg_cache_update = 100; 
	config->len_seg_query_out = 100;
	config->query_width = config->edge_cell_num * config->query_width_rate;
	config->obj_move_speed = 1;
	cout << "Config Initial Completed..." << endl;
}

void CtorBktpond()
{
	hipMallocManaged(&bkt_pond, sizeof(Bktpond));
	bkt_pond->bkt_num = config->bucket_num;
	hipMallocManaged(&(bkt_pond->arr_bkt), bkt_pond->bkt_num * sizeof(Bucket));

	ObjBox *arr_obj;
	hipMallocManaged(&arr_obj, config->bucket_num * config->max_bucket_len * sizeof(ObjBox));
	for (int i = 0; i < config->bucket_num * config->max_bucket_len; i++)
	{
		arr_obj[i].oid = -1;
		arr_obj[i].x = -1;
		arr_obj[i].y = -1;
		arr_obj[i].ts = -1;
	}
	
	for (int i = 0; i < bkt_pond->bkt_num; i++)
	{
		bkt_pond->arr_bkt[i].bid = i;
		bkt_pond->arr_bkt[i].nO = 0;
		bkt_pond->arr_bkt[i].nxt = NULL;
		bkt_pond->arr_bkt[i].arr_obj = arr_obj + i * config->max_bucket_len;
	}
    cout << "Bktpond Initial Completed..." << endl;
}

void CtorBuffer()
{
	hipMallocManaged(&free_bkt_id, sizeof(Buffer));
	free_bkt_id->front = free_bkt_id->rear = 0;
	free_bkt_id->len = config->bucket_num;
	hipMallocManaged(&(free_bkt_id->bkt_id), free_bkt_id->len * sizeof(int));
	for (int i = 0; i < free_bkt_id->len; i++)
		free_bkt_id->push(i);
	cout << "FreeBktId Initial Completed..." << endl;
}

void CtorGridandSecIndex()
{
	hipMallocManaged(&grid, sizeof(Grid));
	grid->cell_num = config->edge_cell_num * config->edge_cell_num;
	hipMallocManaged(&(grid->arr_cell), grid->cell_num * sizeof(Cell));
	for (int i = 0; i < grid->cell_num; i++)
	{
		grid->arr_cell[i].nB = 0;
		grid->arr_cell[i].head = NULL;
	}
	hipMallocManaged(&secindex, sizeof(SecIndex));
	secindex->nI = config->max_obj_num;
	hipMallocManaged(&(secindex->index), secindex->nI * sizeof(SIEntry));
	for (int i = 0; i < secindex->nI; i++)
	{
		secindex->index[i].idx_cell = -1;
		secindex->index[i].idx_bkt = -1;
		secindex->index[i].idx_obj = -1;
	}

	FILE *fp;
	if (config->gaussian_data == 1)
		fp = fopen("generator/gaussian/init.txt", "r");
	else
		fp = fopen("generator/uniform/init.txt", "r");
    if (fp == NULL) 
	{
		cout << "Opening Init.txt Error..." << endl;
		return;
	}
	int oid;
	float x, y, vx, vy, ts;
	for (int i = 0; i < config->max_obj_num; i++)
	{
		fscanf(fp, "%d %f %f %f %f %f\n", &oid, &x, &y, &vx, &vy, &ts);
		int cell_id = Grid::getCellByXY(x, y, config->edge_cell_num);
        if (grid->arr_cell[cell_id].head == NULL)
		{
			grid->arr_cell[cell_id].head = &(bkt_pond->arr_bkt[free_bkt_id->getfront()]);
		    free_bkt_id->pop();
			grid->arr_cell[cell_id].nB++;
		}
		if (grid->arr_cell[cell_id].head->nO >= config->max_bucket_len)
		{
			bkt_pond->arr_bkt[free_bkt_id->getfront()].nxt = grid->arr_cell[cell_id].head;
			grid->arr_cell[cell_id].head = &(bkt_pond->arr_bkt[free_bkt_id->getfront()]);
			free_bkt_id->pop();
			grid->arr_cell[cell_id].nB++;
		}
		grid->arr_cell[cell_id].head->writeObj(oid, x, y, vx, vy, ts);

		secindex->index[oid].idx_cell = cell_id;
		secindex->index[oid].idx_bkt = grid->arr_cell[cell_id].nB - 1;
		secindex->index[oid].idx_obj = grid->arr_cell[cell_id].head->nO - 1;
	}
	fclose(fp);   
	cout << "GridandSecIndex Initial Completed..." << endl;
}

void CtorUpdateBuffer() // 构造一个包含1000万(25段)更新请求的缓冲区; 大小：240M;
{
	hipMallocManaged(&orig_buffer_update, sizeof(UpdateBuffer));
	orig_buffer_update->len = config->block_size_update * 25;
	hipMallocManaged(&(orig_buffer_update->buffer_update), orig_buffer_update->len * sizeof(UpdateType));
	for (int i = 0; i < orig_buffer_update->len; i++)
	{
		orig_buffer_update->buffer_update[i].oid = -1;
		orig_buffer_update->buffer_update[i].x = -1;
		orig_buffer_update->buffer_update[i].y = -1;
		orig_buffer_update->buffer_update[i].vx = -1;
		orig_buffer_update->buffer_update[i].vy = -1;
		orig_buffer_update->buffer_update[i].ts = -1;
	}
	cout << "UpdateBuffer Initial Completed..." << endl;
}

void WriteUpdateBuffer()
{
	UpdateType upd;
	for (int i = 0; i < orig_buffer_update->len; i++)
	{
		fscanf(fp1, "%d %f %f %f %f %f\n", &upd.oid, &upd.x, &upd.y, &upd.vx, &upd.vy, &upd.ts);
		orig_buffer_update->buffer_update[i] = upd;
	}
}
	
void CtorQueryBuffer() // 构造一个包含400万(100段，全部查询请求)的缓冲区；大小：96M;
{
	hipMallocManaged(&orig_buffer_query, sizeof(QueryBuffer));
	orig_buffer_query->len = config->block_size_query * 100;
	hipMallocManaged(&(orig_buffer_query->buffer_query), orig_buffer_query->len * sizeof(QueryType));
    for (int i = 0; i < orig_buffer_query->len; i++)
	{
		orig_buffer_query->buffer_query[i].qid = -1;
		orig_buffer_query->buffer_query[i].minX = -1;
		orig_buffer_query->buffer_query[i].minY = -1;
		orig_buffer_query->buffer_query[i].maxX = -1;
		orig_buffer_query->buffer_query[i].maxY = -1;
		orig_buffer_query->buffer_query[i].ts = -1;
	}
	cout << "QueryBuffer Initial Completed..." << endl;
}

void WriteQueryBuffer()
{
	QueryType que;
	for (int i = 0; i < orig_buffer_query->len; i++)
	{
		fscanf(fp2, "%d %f %f %f %f %f\n", &que.qid, &que.minX, &que.minY, &que.maxX, &que.maxY, &que.ts);
		orig_buffer_query->buffer_query[i] = que;
	}
}

void CtorCacheUpdate()
{
	hipMallocManaged(&cache_update, sizeof(UpdateCacheArea));
	cache_update->cell_num = config->edge_cell_num * config->edge_cell_num;
	cache_update->seg = config->len_seg_cache_update;
	cache_update->len = cache_update->cell_num * cache_update->seg;
	
	hipMallocManaged(&(cache_update->mtx_delete), cache_update->len * sizeof(int));
	hipMallocManaged(&(cache_update->mtx_insert), cache_update->len * sizeof(UpdateType));
	hipMallocManaged(&(cache_update->mtx_fresh), cache_update->len * sizeof(UpdateType));
	for (int i = 0; i < cache_update->len; i++)
	{
		cache_update->mtx_delete[i] = -1;
		
		cache_update->mtx_insert[i].oid = -1;
		cache_update->mtx_insert[i].x = -1;
		cache_update->mtx_insert[i].y = -1;
		cache_update->mtx_insert[i].vx = -1;
		cache_update->mtx_insert[i].vy = -1;
		cache_update->mtx_insert[i].ts = -1;

		cache_update->mtx_fresh[i].oid = -1;
		cache_update->mtx_fresh[i].x = -1;
		cache_update->mtx_fresh[i].y = -1;
		cache_update->mtx_fresh[i].vx = -1;
		cache_update->mtx_fresh[i].vy = -1;
		cache_update->mtx_fresh[i].ts = -1;
	}

	hipMallocManaged(&(cache_update->sum_d), cache_update->cell_num * sizeof(int));
	hipMallocManaged(&(cache_update->sum_i), cache_update->cell_num * sizeof(int));
	hipMallocManaged(&(cache_update->sum_f), cache_update->cell_num * sizeof(int));
	for (int i = 0; i < cache_update->cell_num; i++)
	{
		cache_update->sum_d[i] = 0;
		cache_update->sum_i[i] = 0;
		cache_update->sum_f[i] = 0;
	}
	cout << "CacheUpdate Initial Completed..." << endl;
}

void CtorCacheQuery()
{
	hipMallocManaged(&cache_query, sizeof(QueryCacheArea));
	cache_query->que_num = config->block_size_query;
	int l = (int)config->query_width + 1; // l 为查询区域方形的边长 - 1；
	cache_query->seg1 = (l - 1) * (l - 1);
	cache_query->seg2 = 4 * l;
	cache_query->len1 = cache_query->que_num * cache_query->seg1;
	cache_query->len2 = cache_query->que_num * cache_query->seg2;
	hipMallocManaged(&(cache_query->total), cache_query->len1 * sizeof(int));
	for (int i = 0; i < cache_query->len1; i++) cache_query->total[i] = -1;
	hipMallocManaged(&(cache_query->partial), cache_query->len2 * sizeof(int));
	for (int i = 0; i < cache_query->len2; i++) cache_query->partial[i] = -1;
	hipMallocManaged(&(cache_query->cnt1), cache_query->que_num * sizeof(int));
	for (int i = 0; i < cache_query->que_num; i++) cache_query->cnt1[i] = 0;
	hipMallocManaged(&(cache_query->cnt2), cache_query->que_num * sizeof(int));
	for (int i = 0; i < cache_query->que_num; i++) cache_query->cnt2[i] = 0;
	cout << "CacheQuery Initial Completed..." << endl;
}

void CtorQueryOut()
{
	hipMallocManaged(&query_out, sizeof(QueryOut));
	query_out->query_num = config->block_size_query;
	query_out->seg = config->len_seg_query_out;
	query_out->len = query_out->query_num * query_out->seg;
	hipMallocManaged(&(query_out->obj_id), query_out->len * sizeof(int));
	for (int i = 0; i < query_out->len; i++) query_out->obj_id[i] = -1;
	hipMallocManaged(&(query_out->cnt), query_out->query_num * sizeof(int));
	for (int i = 0; i < query_out->query_num; i++) query_out->cnt[i] = 0;
	cout << "QueryOut Initial Completed..." << endl;
}

void Dtor()
{
    hipFree(config);
	hipFree(bkt_pond);
	hipFree(free_bkt_id);
	hipFree(grid);
	hipFree(secindex);
	hipFree(orig_buffer_update);
	hipFree(orig_buffer_query);
	hipFree(cache_update);  
	hipFree(cache_query);
	hipFree(query_out);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);     
	hipEventDestroy(start1);
	hipEventDestroy(stop1);
    fclose(fp1);
	fclose(fp2);
}


